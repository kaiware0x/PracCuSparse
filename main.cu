#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"

#include "spmv_csr_example.hpp"

static void vecDouble(int *in, int *out, const int n);

int main()
{
    printf("Hello\n");

    const int n = 10;
    int *in = new int[n];
    int *out = new int[n];
    int *answer = new int[n];

    for (int i = 0; i < n; i++)
        in[i] = rand() % 100;
    for (int i = 0; i < n; i++)
        answer[i] = in[i] * 2;

    vecDouble(in, out, n);

    int i;
    for (i = 0; i < n; i++)
    {
        if (answer[i] != out[i])
        {
            printf("error at index = %d\n", i);
            break;
        }
    }
    printf("OK\n");

    delete[] in;
    delete[] out;
    delete[] answer;


    
    ///
    spmv_csr_example();

    return 0;
}

__global__ void kernel_vecDouble(int *in, int *out, const int n)
{
    int i = threadIdx.x;
    if (i < n)
    {
        out[i] = in[i] * 2;
    }
}

static void vecDouble(int *hIn, int *hOut, const int n)
{
    int *dIn;
    int *dOut;
    hipHostMalloc((void **)&dIn, n * sizeof(int));
    hipHostMalloc((void **)&dOut, n * sizeof(int));
    hipMemcpy(dIn, hIn, n * sizeof(int), hipMemcpyHostToDevice);

    kernel_vecDouble<<<1, n>>>(dIn, dOut, n);
    hipDeviceSynchronize();

    hipMemcpy(hOut, dOut, n * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(dIn);
    hipFree(dOut);

}
