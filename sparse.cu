
#include<iostream>
#include<hip/hip_runtime_api.h>
#include<hipblas.h>
#include<hipsparse.h>
#include<hipsparse.h>
#include<thrust/device_vector.h>

const int N = 1024;

int sparse()
{
    /**********************************/
    /********** 入力値の準備 **********/
    /**********************************/

    // CSR形式疎行列のデータ
    //* 要素の値
    //* 列番号
    //* 各行の先頭位置
    double elements[N*3];
    int columnIndeces[N*3];
    int rowOffsets[N+1];

    // 中央差分行列を準備する
    //（対角項が2でその隣が1になる、↓こんなやつ）
    // | 2 1 0 0 0 0 0 0 ・・・ 0 0 0|
    // | 1 2 1 0 0 0 0 0 ・・・ 0 0 0|
    // | 0 1 2 1 0 0 0 0 ・・・ 0 0 0|
    // | 0 0 1 2 1 0 0 0 ・・・ 0 0 0|
    // | 0 0 0 1 2 1 0 0 ・・・ 0 0 0|
    // | 0 0 0 0 1 2 1 0 ・・・ 0 0 0|
    // | 0 0 0 0 0 1 2 1 ・・・ 0 0 0|
    // | 0 0 0 0 0 0 1 2 ・・・ 0 0 0|
    // | 0 0 0 0 0 0 0 0 ・・・ 2 1 0|
    // | 0 0 0 0 0 0 0 0 ・・・ 1 2 1|
    // | 0 0 0 0 0 0 0 0 ・・・ 0 1 2|
    int nonZeroCount = 0;
    rowOffsets[0] = 0;
    for(int i = 0; i < N; i++)
    {
        // 対角項
        elements[nonZeroCount] = 2;
        columnIndeces[nonZeroCount] = i;
        nonZeroCount++;

        // 対角項の左隣
        if(i > 0)
        {
            elements[nonZeroCount] = 1;
            columnIndeces[nonZeroCount] = i - 1;
            nonZeroCount++;
        }

        // 対角項の右隣
        if(i < N-1)
        {
            elements[nonZeroCount] = 1;
            columnIndeces[nonZeroCount] = i + 1;
            nonZeroCount++;
        }

        // 次の行の先頭位置
        rowOffsets[i+1] = nonZeroCount;
    }

    // かけるベクトルを生成
    double vector[N];
    for(int i = 0; i < N; i++)
    {
        vector[i] = i * 0.1;
    }

    // 結果格納ベクトルを生成
    double result[N];

    /**********************************/
    /********** 入力値の転送 **********/
    /**********************************/
    // GPU側の配列を確保
    // （ポインタ管理が面倒なのでthrust使うと便利！）
    thrust::device_vector<double> elementsDevice(N*3);
    thrust::device_vector<int>    columnIndecesDevice(N*3);
    thrust::device_vector<int>    rowOffsetsDevice(N+1);
    thrust::device_vector<double> vectorDevice(N);
    thrust::device_vector<double> resultDevice(N);

    // GPU側配列へ入力値（行列とベクトル）を複製
    thrust::copy_n(elements,      N*3, elementsDevice.begin());
    thrust::copy_n(columnIndeces, N*3, columnIndecesDevice.begin());
    thrust::copy_n(rowOffsets,    N+1, rowOffsetsDevice.begin());
    thrust::copy_n(vector, N, vectorDevice.begin());



    /************************************/
    /********** cuSPARSEの準備 **********/
    /************************************/
    // cuSPARSEハンドルを作成
    ::hipsparseHandle_t cusparse;
    ::hipsparseCreate(&cusparse);

    // 行列形式を作成
    // * 一般的な形式
    // * 番号は0から開始
    ::hipsparseMatDescr_t matDescr;
    ::hipsparseCreateMatDescr(&matDescr);
    ::hipsparseSetMatType(matDescr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    ::hipsparseSetMatIndexBase(matDescr, HIPSPARSE_INDEX_BASE_ZERO);



    /******************************************/
    /********** 行列×ベクトルの計算 **********/
    /******************************************/
    // thrust配列からCUDA用ポインタに変換
    double* elementsPtr   = thrust::raw_pointer_cast(&(elementsDevice[0]));
    int* columnIndecesPtr = thrust::raw_pointer_cast(&(columnIndecesDevice[0]));
    int* rowOffsetsPtr    = thrust::raw_pointer_cast(&(rowOffsetsDevice[0]));
    double* vectorPtr     = thrust::raw_pointer_cast(&(vectorDevice[0]));
    double* resultPtr     = thrust::raw_pointer_cast(&(resultDevice[0]));

    // Csrmv（CSR形式行列とベクトルの積）を実行
    // y = α*Ax + β*y;
    const double ALPHA = 1;
    const double BETA = 0;

    ::hipsparseSpMV(cusparse, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        N, N, nonZeroCount,
        &ALPHA, matDescr, elementsPtr, rowOffsetsPtr, columnIndecesPtr,
        vectorPtr,
        &BETA, resultPtr);

        // Perform SpMV operation with cuSPARSE
float alpha = 1.0f;
float beta = 0.0f;
hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
             &alpha, matA, vecX, &beta, vecY, HIP_R_32F, 
             HIPSPARSE_MV_ALG_DEFAULT, NULL);

    // ::cusparseDcsrmv_v2(cusparse, HIPSPARSE_OPERATION_NON_TRANSPOSE,
    //     N, N, nonZeroCount,
    //     &ALPHA, matDescr, elementsPtr, rowOffsetsPtr, columnIndecesPtr,
    //     vectorPtr,
    //     &BETA, resultPtr);



    /************************************/
    /********** 計算結果を取得 **********/
    /************************************/
    // GPU側配列から結果を複製
    thrust::copy_n(resultDevice.begin(), N, resultDevice);

    // 結果の表示
    for(int i = 0; i < N; i++)
    {
        std::cout << result[i] << std::endl;
    }


    return 0;
}
