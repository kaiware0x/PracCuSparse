#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>

int sparse2()
{
    // Initialize the device
    hipSetDevice(0);

    // Create a cuSPARSE handle
    hipsparseHandle_t handle;
    hipsparseCreate(&handle);

    // Define the matrix dimensions and the vector size
    int num_rows = 5;
    int num_cols = 5;
    int nnz = 13; // number of non-zero elements in the matrix
    int vec_size = 5;

    // Define the matrix and vector on the host
    float *h_matrix = (float *)malloc(nnz * sizeof(float));
    int *h_rowPtr = (int *)malloc((num_rows + 1) * sizeof(int));
    int *h_colInd = (int *)malloc(nnz * sizeof(int));
    float *h_vector = (float *)malloc(vec_size * sizeof(float));

    // Fill the matrix and vector with values...

    // Allocate memory on the device
    float *d_matrix, *d_vector, *d_output;
    int *d_rowPtr, *d_colInd;
    hipMalloc((void **)&d_matrix, nnz * sizeof(float));
    hipMalloc((void **)&d_rowPtr, (num_rows + 1) * sizeof(int));
    hipMalloc((void **)&d_colInd, nnz * sizeof(int));
    hipMalloc((void **)&d_vector, vec_size * sizeof(float));
    hipMalloc((void **)&d_output, num_rows * sizeof(float));

    // Copy the matrix and vector to the device
    hipMemcpy(d_matrix, h_matrix, nnz * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_rowPtr, h_rowPtr, (num_rows + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_colInd, h_colInd, nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_vector, h_vector, vec_size * sizeof(float), hipMemcpyHostToDevice);

    // Perform the matrix-vector multiplication
    hipsparseMatDescr_t descr;
    hipsparseCreateMatDescr(&descr);
    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

    // Create matrix descriptor
    hipsparseSpMatDescr_t matA;
    hipsparseCreateCsr(&matA, num_rows, num_cols, nnz,
                      d_rowPtr, d_colInd, d_matrix,
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);

    // Create vector descriptors
    hipsparseDnVecDescr_t vecX, vecY;
    hipsparseCreateDnVec(&vecX, num_cols, d_vector, HIP_R_32F);
    hipsparseCreateDnVec(&vecY, num_rows, d_output, HIP_R_32F);

    float alpha = 1.0;
    float beta = 0.0;
    hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                 &alpha, matA, vecX, &beta, vecY, HIP_R_32F,
                 HIPSPARSE_SPMV_ALG_DEFAULT, NULL);

    // Copy the result back to the host
    float *h_output = (float *)malloc(num_rows * sizeof(float));
    hipMemcpy(h_output, d_output, num_rows * sizeof(float), hipMemcpyDeviceToHost);

    // Clean up
    free(h_matrix);
    free(h_rowPtr);
    free(h_colInd);
    free(h_vector);
    free(h_output);
    hipFree(d_matrix);
    hipFree(d_rowPtr);
    hipFree(d_colInd);
    hipFree(d_vector);
    hipFree(d_output);
    hipsparseDestroy(handle);

    return 0;
}
